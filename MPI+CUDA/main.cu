﻿//MPIv2 + CUDAv2方法
#include <mpi.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <time.h>
#include <thread>
#include <math.h>

using namespace std;
#define BLOCK_SIZE 16
#define error 0.2
#include"cudaMatrixMul.cuh"
#include <chrono>


//void printTime() {
//	auto now = std::chrono::system_clock::now();
//	//通过不同精度获取相差的毫秒数
//	uint64_t dis_millseconds = std::chrono::duration_cast<std::chrono::milliseconds>(now.time_since_epoch()).count()
//		- std::chrono::duration_cast<std::chrono::seconds>(now.time_since_epoch()).count() * 1000;
//	time_t tt = std::chrono::system_clock::to_time_t(now);
//	auto time_tm = localtime(&tt);
//	char strTime[25] = { 0 };
//	sprintf(strTime, "%d-%02d-%02d %02d:%02d:%02d %03d", time_tm->tm_year + 1900,
//		time_tm->tm_mon + 1, time_tm->tm_mday, time_tm->tm_hour,
//		time_tm->tm_min, time_tm->tm_sec, (int)dis_millseconds);
//	cout << strTime << endl;
//}


int main(int argc, char** argv) {
	int mmm, nnn, kkk;
	mmm = 1024;
	nnn = 1024;
	kkk = 1024;
	if (argc >= 4) {
		mmm = atoi(argv[1]);
		nnn = atoi(argv[2]);
		kkk = atoi(argv[3]);
	}
	if (argc == 2) {
		mmm = atoi(argv[1]);
		nnn = atoi(argv[1]);
		kkk = atoi(argv[1]);
	}
	// MPI初始化
	int rank = 0, numprocs = 0;
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);  //当前进程号
	MPI_Comm_size(MPI_COMM_WORLD, &numprocs);  //进程个数
	MPI_Status status;

	// 初始化数据
	Matrix A, B, C, buffer, ans;
	matrixCreate(A, mmm, nnn);
	matrixCreate(B, nnn, kkk);
	matrixCreate(C, mmm, kkk);

	int line = mmm / numprocs;
	// 缓存大小大于等于要处理的数据大小，大于时只需关注实际数据那部分
	matrixCreate(buffer, line, nnn);
	matrixCreate(ans, line, kkk);

	int i, j, k;
	// 主处理器
	if (rank == 0) {
		printf("order of matrix: %d-%d-%d\n", mmm, nnn, kkk);
		printf("num procs: %d\n", numprocs);
		// 将矩阵A和B随机赋值
		initMatrix(A);
		initMatrix(B);

		double start = MPI_Wtime(), stop, time_MPI, s1, s2, tn;
		// 将矩阵B发送给其他进程
		// 1 数据广播
		s1 = MPI_Wtime();
		MPI_Bcast(B.elements, B.height * B.width, MPI_FLOAT, 0, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);
		s2 = MPI_Wtime();
		cout << "数据广播用时 :" << (s2 - s1) * 1000 << endl;

		// 将矩阵A的各行发送给各个从进程
		// 2 数据分发
		s1 = MPI_Wtime();
		MPI_Scatter(A.elements, line * A.width, MPI_FLOAT, buffer.elements, line * nnn, MPI_FLOAT, 0, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);
		s2 = MPI_Wtime();
		cout << "数据分发用时:" << (s2 - s1) * 1000 << endl;

		// 3 计算本地结果
		s1 = MPI_Wtime();
		matrixMulCudaV2(buffer, B, ans, 1, 0);
		MPI_Barrier(MPI_COMM_WORLD);
		s2 = MPI_Wtime();
		printf("计算用时:%f\n", (s2-s1)*1000);

		// 4 结果聚集
		s1 = MPI_Wtime();
		MPI_Gather(ans.elements, line * kkk, MPI_FLOAT, C.elements, line * kkk, MPI_FLOAT, 0, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);
		s2 = MPI_Wtime();
		printf("结果聚集用时:%f\n", (s2 - s1) * 1000);

		// 5 剩余行处理（处理不能整除的情况）
		float temp;
		int rest = mmm % numprocs;
		if (rest != 0) {
			cout << "进行剩余行处理，剩余行：" << rest << endl;
			for (i = mmm - rest; i < mmm; i++)
				for (j = 0; j < kkk; j++) {
					temp = 0.0f;
					for (k = 0; k < nnn; k++)
						temp += A.elements[i * nnn + k] * B.elements[k * kkk + j];
					C.elements[i * kkk + j] = temp;
				}
		}

		stop = MPI_Wtime();
		time_MPI = stop - start;

		// 依据单线程计算结果为标准验证MPI计算结果是否正确
		// 一般来大规模矩阵相乘计算时间较长，方阵阶数到2000以上时会取消单线程结果验证
		double time_single = 0.0;
		Matrix C_single;
		matrixCreate(C_single, mmm, kkk);
		start = MPI_Wtime();
		matrixMulSingle(A, B, C_single);
		stop = MPI_Wtime();
		time_single = (stop - start);

		// 输出结果
		cout << "--single method runtime: \t " << time_single * 1000 << "ms" << endl;
		cout << "--MPIv2 + CUDAv2  runtime: \t" << time_MPI * 1000 << "ms\t" <<
			"speedUP:" << time_single / time_MPI << "\t  " <<
			"maximum error: " << maxErrorOfMatrixs(C_single, C) << "\t" << endl;

		/*printMatrix(C_single);
		printMatrix(C);*/

	}
	// 从进程接收数据，计算结果并发送给主进程
	else {
		// 数据广播
		MPI_Bcast(B.elements, B.width * B.height, MPI_FLOAT, 0, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);

		// 数据分发
		MPI_Scatter(A.elements, line * nnn, MPI_FLOAT, buffer.elements, line * nnn, MPI_FLOAT, 0, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);

		// 计算本地
		matrixMulCudaV2(buffer, B, ans, 1, 0);
		MPI_Barrier(MPI_COMM_WORLD);

		// 结果聚集
		MPI_Gather(ans.elements, line * kkk, MPI_FLOAT, C.elements, line * kkk, MPI_FLOAT, 0, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);
	}

	MPI_Finalize();//结束
	return 0;
}