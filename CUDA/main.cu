#include "hip/hip_runtime.h"
﻿#include"CudaMatrixMulMethod.cuh"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

#define BLOCK_SIZE 16
#define error 0.2

int main(int argc, char** argv) {
	//矩阵尺寸
	int mmm, nnn, kkk;
	mmm = 512;
	nnn = 512;
	kkk = 512;
	if (argc >= 4) {
		mmm = atoi(argv[1]);
		nnn = atoi(argv[2]);
		kkk = atoi(argv[3]);
	}

	printf("Order of matrix: %d-%d-%d\n", mmm, nnn, kkk);
	printf("Maximum allowable error of this operation: %f\n", error);
	Matrix A, B, C_single, C_cudaV1, C_cudaV2;
	float start, end;
	float time_cuda_V1, time_cuda_V2, time_single;  //用时
	matrixCreateAndInit(A, mmm, nnn);
	matrixCreateAndInit(B, nnn, kkk);
	matrixCreate(C_single, mmm, kkk);
	matrixCreate(C_cudaV1, mmm, kkk);
	matrixCreate(C_cudaV2, mmm, kkk);

	// 单线程方法
	start = clock();
	matrixMulSingle(A, B, C_single);
	end = clock();
	time_single = end - start;
	cout << "Single thread runtime: \t" << time_single << "ms\t" << endl;

	// Cuda方法V1
	time_cuda_V1 = matrixMulCudaV1(A, B, C_cudaV1, 100);
	cout << "Cuda method 1 runtime: \t" << time_cuda_V1 << "ms\t" <<//运行时长
		"speedUP:" << time_single / time_cuda_V1 << "\t  " <<//加速比
		"maximum error: " << maxErrorOfMatrixs(C_single, C_cudaV1) << "\t";//最大误差
	if (matrixCompare(C_single, C_cudaV1, error)) {//在误差允许范围结果是否正确
		cout << "Result Correct" << endl;
	}
	else { cout << "Result Error" << endl; }

	// Cuda方法V2
	time_cuda_V2 = matrixMulCudaV2(A, B, C_cudaV2, 100);
	cout << "Cuda method 2 runtime: \t" << time_cuda_V2 << "ms\t" <<
		"speedUP:" << time_single / time_cuda_V2 << "\t  " <<
		"maximum error: " << maxErrorOfMatrixs(C_single, C_cudaV2) << "\t";
	if (matrixCompare(C_single, C_cudaV2, error)) {
		cout << "Result Correct" << endl;
	}
	else { cout << "Result Error" << endl; }

	//printMatrix(C_cudaV1);
	//printMatrix(C_cudaV2);
	cout << "ending..." << endl;
}